#include "hip/hip_runtime.h"
/*
 *
 * Copyright 2015, Google Inc.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are
 * met:
 *
 *     * Redistributions of source code must retain the above copyright
 * notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above
 * copyright notice, this list of conditions and the following disclaimer
 * in the documentation and/or other materials provided with the
 * distribution.
 *     * Neither the name of Google Inc. nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR
 * A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
 * OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
 * SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
 * LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
 * DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
 * THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 */
 
#define QTYPE_float
#define QTYPE_ACC_float
//#define QTYPE_short
//#define QTYPE_ACC_short
//#define QTYPE_half
//#define QTYPE_ACC_half

#include "ReadUntilClient.h"
#include "flash_dtw.cuh"
#include "dtw.hpp"
#include "segmentation.hpp"
// #include "cuda_utils.h" // CUERR()

#if defined(_WIN32)
	#include <conio.h>
	#include <windows.h>
	extern "C"{
		#include "getopt.h"
	}
	#include <direct.h>
	#define GetCurrentDir _getcwd
	#define FILE_SEPARATOR "\\"
#else
	#include <unistd.h>
	#define GetCurrentDir getcwd
	#define FILE_SEPARATOR "/"
#endif

#include <signal.h>

// bool hasEnding (std::string const &fullString, std::string const &ending) {
    // if (fullString.length() >= ending.length()) {
        // return (0 == fullString.compare (fullString.length() - ending.length(), ending.length(), ending));
    // } else {
        // return false;
    // }
// }

std::ofstream log_file;
void handler(int s){
	printf("Caught signal %d. Closing log\n",s);
	log_file << "Caught signal " << s << std::endl;
	log_file.close();
	exit(1); 	
}

int flash_dtw_callback(QTYPE* query_values, unsigned long long int num_query_values, char* query_name, float colDev, float maxP, float maxFDR, float maxRanks, float znorm, match_record** results, float miniDtwSize, float miniDtwWarp, bool record_match_anchors, float useFastCalc, bool use_std, int use_hard_dtw){
	int num_results = 0;
	flash_dtw(query_values, num_query_values, query_name, colDev, maxP, maxFDR, maxRanks, znorm, results, &num_results, miniDtwSize, miniDtwWarp, record_match_anchors, useFastCalc, use_std, use_hard_dtw);
	return num_results;
}

template<typename T>
void adaptive_segmentation_callback(T **sequences, size_t *seq_lengths, int num_seqs, int min_segment_length, T ***segmented_sequences, size_t **segmented_seq_lengths){
	adaptive_segmentation(sequences, seq_lengths, num_seqs, min_segment_length, segmented_sequences, segmented_seq_lengths);
}

template<class T>
void normalize_queries_callback(T *query_values, size_t query_length, int normalization_mode, float* welford_mean, float* welford_ssq, long total_values_znormalized){
	load_and_normalize_queries(query_values, query_length, normalization_mode, welford_mean, welford_ssq, total_values_znormalized);
}

void free_results_callback(match_record** results){
	hipHostFree(*results);
}

int main(int argc, char** argv) {
	
	int verbose = 0;
	int help = 0;
	
	int signal_type = MEAN_SIGNAL;
	int rna = 0;
	int complement_only = 0;
	int complement = 0;
	
	int use_directory_as_subject = 0;

	int bed_file = 0;
	char* bed_filename;
	
	char *std_filename;
	
	int znorm = 0;
	int use_fast_anchor_calc = 0;
	bool use_std = false;
	int use_hard_dtw = 1;
	int minidtw_size = 10; // matching block size
	int minidtw_warp = 2;
	
	int match_max_ranks = 100; // Up to how many results do you want to show that meet the FDR and p value criteria?  Cannot be more than 2^20 (~1M)
	float match_max_pvalue = 0.01;
	float match_max_fdr = 1;
	float max_collinearity_dev = 0.25;
	int min_segment_length_sub = 0; // nanopore RNA segment length by default for the subject
	int min_segment_length_query = 0; // nanopore RNA segment length by default for the query
	
	std::string host = "localhost"; // Default minknow host
	int port = 8000; // Default minknow port
	int channel_begin = 1;
	int channel_end = 512;
	int pore_buff_size = 24000;
	int upper = 1300;
	int lower = 300;
	
	int instrand = 3;

	int num_devices;
	hipGetDeviceCount(&num_devices);
	int num_threads = num_devices;
	int num_chan_threads = 1;
	
	// int fasta_file = 0;
	// char* fasta_filename;
	// int single_strand = 0;
	
	// int prefix = 0;
	// int minimal = 0;
	// int requested_suffix_length = 100;
	// int reverse = 0;

	// int input_is_binary = 0;
	int selection = 0; // 0 for positive, 1 for negative selection

	char c;

	char buff[FILENAME_MAX];
	GetCurrentDir( buff, FILENAME_MAX );
	std::string log_output_dir(buff);
	log_output_dir += "/log";

	while( ( c = getopt (argc, argv, "S:p:f:r:m:w:q:s:q:d:b:z:H:P:B:E:g:u:l:t:T:O:i:cCRnFDNvh") ) != -1 ) {
		switch(c) {		
			case 'S':
				if(optarg) use_directory_as_subject = atoi(optarg);
				break;
			case 'p':
				if(optarg) match_max_pvalue = atof(optarg);
				break;
			case 'f':
				if(optarg) match_max_fdr = atof(optarg);
				break;
			case 'r':
				if(optarg) match_max_ranks = atoi(optarg);
				break;
			case 'm':
				if(optarg) max_collinearity_dev = atof(optarg);
				break;
			case 'w':
				if(optarg) minidtw_warp = atoi(optarg);
				break;
			case 'a':
				if(optarg) minidtw_size = atoi(optarg);
				break;
			case 's':
				if(optarg) min_segment_length_sub = atoi(optarg);
				break;
			case 'q':
				if(optarg) min_segment_length_query = atoi(optarg);
				break;
			case 'd':
				if(optarg) std_filename = optarg;
				use_std = 1;
				break;
			case 'b':
				if(optarg) bed_filename = optarg;
				bed_file = 1;
				break;
			case 'z':
				if(optarg) znorm = atoi(optarg);
				break;
			case 'H':
				if(optarg) host = optarg;
				break;
			case 'P':
				if(optarg) port = atoi(optarg);
				break;
			case 'B':
				if(optarg) channel_begin = atoi(optarg); 
				break;
			case 'E':
				if(optarg) channel_end = atoi(optarg); 
				break;
			case 'g':
				if(optarg) pore_buff_size = atoi(optarg); 
				break;
			case 'u':
				if(optarg) upper = atoi(optarg); 
				break;
			case 'l':
				if(optarg) lower = atoi(optarg); 
				break;
			case 't':
				if(optarg) num_devices = atoi(optarg);
				if(num_devices > num_threads){
					std::cerr << "Number of threads specified (" << num_devices << ") exceeds number of GPUs on this system. Defaulting to " << num_threads << std::endl;
				}
				break;
			case 'T':
				if(optarg) num_chan_threads = atoi(optarg);
				break;
			// case 'A':
				// if(optarg) fasta_filename = optarg;
				// fasta_file = 1;
				// break;
			// case 'L':
				// requested_suffix_length = atoi(optarg);
				// break;
			case 'O':
				if(optarg) log_output_dir = optarg;
				break;
			case 'i':
				instrand = atoi(optarg);
				break;
			case 'c':
				complement = 1;
				break;
			case 'C':
				complement_only = 0;
				break;
			case 'R':
				rna = 1;
				break;
			case 'n':
				signal_type = STDDEV_SIGNAL;
				break;
			case 'F':
				use_fast_anchor_calc = 1;
				break;
			case 'D':
				use_hard_dtw = 0;
				break;
			// case 'M':
				// minimal = 1;
				// break;
			// case 'R':
				// reverse = 1;
				// break;
			// case 'y':
				// prefix = 1;
				// break;
			// case 'o':
				// single_strand = 1;
				// break;
			// case 'i':
				// input_is_binary = 1;
				// break;
			case 'N':
				selection = 1;
				break;
			case 'v':
				verbose = 1;
				break;
			case 'h':
				help = 1;
				break;	
			default:
				/* You won't actually get here. */
				break;
		}
	}
	
	int num_args = argc - optind;
	if (help || 
		num_args != 1){
		std::cerr << "Usage: " << argv[0] << " [options] <reference_genome>" << std::endl
					<< "Client that receives reads from a MinION device through MinKOWN and applies NVIDIA GPU (CUDA-enabled) accelerated Dynamic Time Warping on them." << std::endl
					<< "DTW implementation based on the FLASH DTW anchor co-linearity method." << std::endl
					<< "reference_genome is a subject file that's been indexed with magenta_short_index, which will be subject-scaled and cast to " << QTYPE_NAME << ")" << std::endl
					<< "Options for reading BED file:" << std::endl
					<< "[-b Read in BED file to compare matches to. Must provide file path with this option]" << std::endl << std::endl
					<< "Options relating to connecting and reading in values are:" << std::endl
					<< "[-H Host to open a connection on] default=" << host << std::endl
					<< "[-P Port to connect to] default=" << port << std::endl
					<< "[-B Start channel to get reads from] default=" << channel_begin << std::endl
					<< "[-E End channel to get reads from] default=" << channel_end << std::endl 
					<< "[-g Size of the buffers that will be used to store reads for each pore] default=" << pore_buff_size << std::endl
					<< "[-u Upper limit to check for which determines if a read is instrand] default=" << upper << std::endl
					<< "[-l Lower limit to check for which determines if a read is instrand] default=" << lower << std::endl
					<< "[-i The value in the subject file that states when data was in strand.] default= " << instrand << std::endl
					<< "[-t Number of threads reads will be processed on] default=" << num_threads << std::endl
					<< "[-T Number of threads to send and receive data to and from the MinKNOW] default=" << num_chan_threads << std::endl << std::endl
					<< "Options related to matches are:" << std::endl
					// << "[-i Input is binary]" << std::endl
					<< "[-p P-value limit for reporting matches (i.e. anchor DTW distance Mann-Whitney test random match probability for the subject DB used, floating point)] default=" << match_max_pvalue << std::endl
					<< "[-f FDR limit for reporting matches (i.e. Benjamini-Hochberg multiple-testing corrected p-value for the subject DB used, floating point)] default=" << match_max_fdr << std::endl
					<< "[-r ranks limit for reporting matches (i.e. the number of matches to report that pass the pvalue and FDR criteria. Guaranteed to be the best matches unless # passing criteria are > 2^20)] default=" << match_max_ranks << std::endl
					<< "[-m Warp max (proportion of length deviation allowed between query and subject in alignment, larger=more sensitive & longer runtime, floating point)> default=" << max_collinearity_dev << std::endl
					<< "[-s Minimum segment length subject (defines the minimum segment length that will be accepted for the subject. segmentation will not run if set to 0, int)] default=" << min_segment_length_sub << std::endl
					<< "[-q Minimum segment length query (defines the minimum segment length that will be accepted for the query. segmentation will not run if set to 0, int)] default=" << min_segment_length_query << std::endl << std::endl
					<< "[-N Negative selection (default is positive)]" << std::endl 
					<< "[-z Normalization type for z-norm of query against subject: 0 = NO_ZNORM, 1 = LOCAL_ZNORM, 2 = ONLINE_ZNORM, 3 = GLOBAL_ZNORM, 4 = PERCENTILE_ZNORM] default=" << znorm << std::endl
					<< "[-F Enable fast non-colinear-distances sampling algorithm (by default uses thorough sampling algorithm)]" << std::endl
					<< "[-d Enable standard deviation distance calculation (requires subject_std file as additional input)]" << std::endl
					<< "[-D Disable hard_dtw so that soft_dtw of size 10 and warp 2 may be used ('-M 10 -W 2' arguments)]" << std::endl
					<< "[-a Mini-DTW size, length of comparison between query and subject within a mini-DTW iteration] default=" << minidtw_size << std::endl
					<< "[-w Mini-DTW warp, boundary left and right of the diagonal in a mini-DTW matrix that the path may traverse] default=" << minidtw_warp << std::endl << std::endl
					// << "Options related to FastA indexing are" << std::endl
					// << "[-A index FastA file. Must provide file path with this option. NOTE: reference_genome argument would be the FastA file here]" << std::endl
					// << "[-o single strand indexing only]" << std::endl
					<< "[-n Use standard deviation for signal type in FastA files. Default uses mean.]" << std::endl
					<< "[-c also generate signal for the reverse complement strand]" << std::endl
					<< "[-C exclude default forward strand encoding]" << std::endl
					<< "[-R convert input as RNA (default is DNA)]" << std::endl << std::endl
					// << "Options related to FastA prefixing are" << std::endl
					// << "[-y prefix FastA sequences]" << std::endl
					// << "[-R reverse the sequence (e.g. for nanopore 3'->5' direct RNA analysis)]" << std::endl
					// << "[-L unique suffix length to include in the output (default " << requested_suffix_length << ")]" << std::endl
					// << "[-M minimal output]" << std::endl
					// << "Note: these options are only used when giving a FastA file as a reference. They will do nothing otherwise." << std::endl << std::endl
					<< "Options for logging are:" << std::endl
					<< "[-O Output directory for logs. Filename will be: [Year][Month][Day]_[Time]_ont_log.txt default directory is current working directory (" << log_output_dir << ")" << std::endl << std::endl
					<< "Additional options are:" << std::endl
					<< "[-v verbose mode]" << std::endl
					<< "[-h help (this message)]" << std::endl
					<< "Note: Reads will be obtained from the start channel to the end channel. End channel must not be smaller than start channel" << std::endl << std::endl;

		if(num_args > 1) std::cerr << "Error: Too many arguments." << std::endl;
		if(num_args < 1) std::cerr << "Error: No arguments given." << std::endl;

		return 0;
	}

	if(verbose) std::cerr << "Running in verbose mode:" << std::endl;
	ITree::interval_vector bed_intervals;
	if(bed_file && !populateITree(bed_intervals, bed_filename, verbose)){
		std::cerr << "Unable to populate ITree. Exiting." << std::endl;
		return 0;
	}

	time_t now = time(0);
    struct tm *tstruct = localtime(&now);

	int year = 1900 + tstruct->tm_year;
	int month = 1 + tstruct->tm_mon;
	int day = tstruct->tm_mday;
	int hour = 1 + tstruct->tm_hour;
	int minute = 1 + tstruct->tm_min;
	int second = 1 + tstruct->tm_sec;

	std::string log_output_name = std::to_string(year) + std::to_string(month) + std::to_string(day) + "_" + std::to_string(hour) + "-" + std::to_string(minute) + "-" + std::to_string(second) + "_ont_log.txt";

	std::string log_file_path = hasEnding(log_output_dir, FILE_SEPARATOR) ? log_output_dir + log_output_name : log_output_dir + FILE_SEPARATOR + log_output_name;
	if(verbose) std::cerr << "Output log path will be: " << log_file_path << std::endl;

	log_file.open(log_file_path, std::ios::out);
	if(verbose) std::cerr << "Running with " << num_chan_threads << " threads for sending/ receiving data and " << num_threads << " threads for running DTW." << std::endl;
	log_file << "Starting log: " << std::endl;
	signal(SIGINT, handler);

	// char *ref_filename;
	// if(!fasta_file){
		// ref_filename = argv[optind];
	// }

	// QTYPE *subject_values;
	// // Allow really big files
	// unsigned long long int num_subject_values = 0;
	// QTYPE *subject_stds;
	// unsigned long long int num_subject_stds;
	// if(!use_std)
		// subject_stds = 0;

	// std::vector< std::pair<size_t, char *> > subject_offsets;
	// subject_offsets.reserve(20);
	
	// Slurp the subject file up all at once. *subject_values will be dynamically allocated for us (free it ASAP please)
	// if(fasta_file){
		// if(!complement && complement_only){
			// std::cerr << "No strand was selected for encoding (i.e. both -c and -C, specified) aborting.\n";
			// return 2;
		// }
		// if(prefix){
			// std::vector<std::string> seq_prefixes;
			// if(!fasta2prefixes(fasta_filename, seq_prefixes, reverse, requested_suffix_length, minimal, verbose)){
				// std::cerr << "Unable to prefix sequences from " << fasta_filename << " so exiting." << std::endl;
				// return 0;
			// }
			// if(!populateSubjectWithFastAPrefix(&subject_values, &num_subject_values, seq_prefixes, subject_offsets, signal_type, rna, complement, complement_only, verbose)){
				// std::cerr << "Unable to populate subject with FastA prefixes. Aborting." << std::endl;
				// return 0;
			// }
		// }else{
			// if(!populateSubjectWithFastA(&subject_values, &num_subject_values, fasta_filename, subject_offsets, signal_type, rna, complement, complement_only, verbose)){
				// std::cerr << "Unable to populate subject with FastA file. Aborting." << std::endl;
				// exit(1);
			// }
		// }
	// }
	// else{
		// if(verbose) std::cerr << "Reading subject from file: " << ref_filename << std::endl;
		// std::string rawfile_name(std::string(ref_filename)+std::string(".hpr"));
		
		// if(read_data<QTYPE>(stringToChar(ref_filename), &subject_values, &num_subject_values)){
			// std::cerr << "Error while reading subject file, aborting" << std::endl;
			// return 3;
		// }
		// if(bed_file){
			// if(!load_subject_index(ref_filename, subject_offsets, &single_strand)){
				// std::cerr << "Could not load subject index for " << ref_filename << ", aborting." << std::endl;
				// exit(1);
			// }
		// }
		// // free(rawfile_name_c);
	// }
	
	char** ref_filenames;
	int num_files = 0;
	if(use_directory_as_subject){
		num_files = getAllFilesFromDir(argv[optind], &ref_filenames);
		if(num_files == 0){
			std::cerr << "Could not get any compatible files from " << argv[optind] << " so aborting." << std::endl;
			return 0;
		}
	} else{	// Only one file used as a subject
		ref_filenames = (char**)malloc(sizeof(char*));
		*ref_filenames = argv[optind];
		num_files = 1;
	}
	
	short strand_flags = 0;
	if(complement){
		strand_flags |= COMPLEMENT_STRAND;
	}
	if(!complement_only){
		strand_flags |= FORWARD_STRAND;
	}
	
	if(verbose) std::cerr << "Initializing GPU device" << std::endl;
	hipSetDevice(0);									CUERR("Setting GPU device to be used");
	hipDeviceReset();									CUERR("Resetting GPU device");
	
	QTYPE **subject_values = 0;
	size_t *num_subject_values = 0;
	QTYPE **subject_stds = 0;
	size_t *num_subject_stds = 0;
	char **sequence_names = 0;
	if(!use_std){
		hipHostMalloc(&subject_stds, sizeof(QTYPE*));	CUERR("Allocating memory for subject stds array");
		*subject_stds = 0;
	}
	std::vector< std::pair<size_t, char*> > subject_offsets;
	subject_offsets.reserve(20);
	
	// Slurp the subject file up all at once. *subject_values will be dynamically allocated for us (free it ASAP please)
	//------Get subject data------
	if(verbose) std::cerr << "Reading subject data from " << *ref_filenames << std::endl;
	int num_sequences_sub = read_data<QTYPE>(ref_filenames, num_files, &subject_values, &sequence_names, &num_subject_values, instrand, rna, signal_type, strand_flags);
	
	if(use_std) {
		if(verbose){
			std::cerr << "Reading subject_std from file " << std_filename << std::endl;
		}
		if(read_data<QTYPE>(&std_filename, 1, &subject_stds, &sequence_names, &num_subject_stds, 0, 0, 0, 0) == 0){
			std::cerr << "Error while reading subject std file, aborting" << std::endl;
			return 3;
		}
		if(*num_subject_values != *num_subject_stds) {
			std::cerr << "Error number of subject values does not match number of subject standard deviations, aborting" << std::endl;
			return 3;
		}
	} 

	size_t subject_length = 0;
	QTYPE* merged_subject = 0; 
	if(min_segment_length_sub > 0){
		QTYPE **segmented_sequences = 0;
		size_t *segmented_seq_lengths = 0;
		adaptive_segmentation<QTYPE>(subject_values, num_subject_values, num_sequences_sub, min_segment_length_sub, &segmented_sequences, &segmented_seq_lengths);
		merged_subject = merge_data(segmented_sequences, segmented_seq_lengths, subject_offsets, sequence_names, num_sequences_sub, &subject_length);
		
		if(verbose){
		  std::cerr << "Finished segmenting subject into " << subject_length << " values" << std::endl;
		  std::cerr << "Loading subject (" << subject_length << " values) to GPU" << std::endl;
		}
		std::cerr << std::endl;

		// hipHostFree(segmented_sequences);
		// hipHostFree(segmented_seq_lengths);
	} else{
		merged_subject = merge_data(subject_values, num_subject_values, subject_offsets, sequence_names, num_sequences_sub, &subject_length);
		if(verbose){
			std::cerr << "Loading subject (" << subject_length << " values) to GPU" << std::endl;
		}
		std::cerr << std::endl;
	}
	if(verbose) std::cerr << "Loading subject (" << num_subject_values << " values) to GPU" << std::endl;
	load_subject(merged_subject, *subject_stds, subject_length, use_std);
	
	// load_subject(subject_values, subject_stds, num_subject_values, use_std);
	// free(subject_values);
	// free(subject_stds);

	// Create a new connection
	// Connection new_con(host, port, verbose);

	// Create a client for the MinKNOW
	ReadUntilClient client(host, port, pore_buff_size, upper, lower, channel_begin, channel_end, verbose);
	
	flash_callback flash_callback_ptr= &flash_dtw_callback;
	aSegmentation_callback<QTYPE> adaptive_segmentation_ptr = &adaptive_segmentation_callback;
	LoadNormalizeQueries_callback<QTYPE> normalize_queries_ptr = &normalize_queries_callback;
	FreeResults_callback free_results_ptr = &free_results_callback;

	std::cerr << "Reads on " << num_threads << " threads:" << std::endl;
	client.ReadsRequest(flash_callback_ptr, adaptive_segmentation_ptr, normalize_queries_ptr, free_results_ptr, num_threads, num_chan_threads, 
									min_segment_length_query, max_collinearity_dev, match_max_pvalue, match_max_fdr, match_max_ranks, subject_offsets, 
									bed_intervals, selection, use_fast_anchor_calc, znorm, use_std, minidtw_size, minidtw_warp, use_hard_dtw, log_file, verbose);

	// Dealocate after running
	client.DeallocatePoreBuff(verbose);
	log_file.close();

	return 0;
}
